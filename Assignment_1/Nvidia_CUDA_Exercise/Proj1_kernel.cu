﻿// ELEC 522 Project 1 CUDA Code
// Adapted from Nvidia learning resources and web site
// Note that there is a lot of error checking on GPU setup

// Include files for Helper functions and utilities to work with CUDA
#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>

// Regular C include file
#include <stdio.h>


// CPU Helper code declaration for CPU function to be detailed later in the file
hipError_t addWithCuda(int *c, int *a, int *b, unsigned int size);

// CUDA definition for code to actually run on GPU
// Inputs are a and b, result is written into c
// Note that there is no loop as would be on the CPU code
// Simple one dimensional "threadIdx.x" when called sets parallelism
__global__ void addKernel(int *c, int *a, int *b)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    c[i] = a[i] + b[i];
}
// CUDA definition ends
 

// C Code to run on main CPU and then call GPU kernel
int main()
{
    const int arraySize = 64000;
    int a[arraySize] = { 0 };
    int b[arraySize] = { 0 };
    int c[arraySize] = { 0 };
    int i;

    for (i = 0; i < arraySize; i++) {
        a[i] = { i + 1 };
        b[i] = { (i + 1) * 10 };
    }

    // Add vectors in parallel. Call CPU helper function
    // This helper then calls GPU
    hipError_t cudaStatus;
    cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // After results are returned by GPU, the CPU prints results returned in c vector.
    printf("Vector length is %d\n", arraySize);
    printf("First 8 values {1,2,3,4,5,6,7,8} + {10,20,30,40,50,60,70,80} = {%d,%d,%d,%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7]);
    printf("Final 2 values of vector {%d,%d} + {%d,%d} = {%d,%d}\n",
        a[arraySize - 2], a[arraySize - 1], b[arraySize - 2], b[arraySize - 1], c[arraySize - 2], c[arraySize - 1]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
// End of main CPU function.


// CPU Helper function for using CUDA to add vectors in parallel.
// The call to the GPU will happen in this helper.
hipError_t addWithCuda(int *c, int *a, int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;


    // Parameters set to execute the kernel on GPU
    // Do one iteration of the code. More iterations would give better average time results
    int nIter = 1;
    // Number of parallel threads to execute on the GPU. Here it is the vector length or size.
    // Maximum number of threads per block is 1024.
    int threads = 1016;
    if (threads > 1024)
        fprintf(stderr, "Maximum threads/block is 1024. Increase block size and reduce threads/block. \n\n");
    // Number of blocks on the GPU. Threads can be grouped into blocks for control.
    // For small problem one block is sufficient. If more than 1024 threads increase blocks.
    int blocks = 63;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        exit(EXIT_FAILURE);
    }

    // Allocate GPU buffers for three vectors (two input, one output) 
    // These three buffers on GPU will be where we transfer data from CPU to GPU to do calculation.
    // cudaMalloc allocates space on GPU DRAM
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        exit(EXIT_FAILURE);
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        exit(EXIT_FAILURE);
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        exit(EXIT_FAILURE);
    }

    // Copy input vectors from host memory to GPU buffers.
    // cudaMemcpy is the transfer function and "cudaMemcpyHostToDevice" is the direction.
    // This example uses explicit allocation and copy to see the details in first programs.
    // The "Unified Memory" model would "hide" these details from the programmer.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        exit(EXIT_FAILURE);
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        exit(EXIT_FAILURE);
    }

    // Allocate CUDA start and stop events that we'll use for timing analysis
    hipEvent_t start;
    cudaStatus = hipEventCreate(&start);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    cudaStatus = hipEventCreate(&stop);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    // Record the start event timestamp on the GPU clock
    cudaStatus = hipEventRecord(start, NULL);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    // Launch a kernel on the GPU with one thread for each element.
    // This is the actual driver call to the GPU setting the parallelism.
    // blocks and threads set the parallelism; dev_a and dev_b data as already been copied.
    // Result will be written in to dev_c but will remain in GPU until copied back later.
    addKernel<<<blocks, threads>>>(dev_c, dev_a, dev_b);


   // Record the stop event on GPU after "addKernel" finishes.
    cudaStatus = hipEventRecord(stop, NULL);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete. This barrier is needed before reading back data.
    cudaStatus = hipEventSynchronize(stop);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        exit(EXIT_FAILURE);
    }

    // Compute GPU kernel elapsed time as difference of stop and start timestamps.
    float msecTotal = 0.0f;
    cudaStatus = hipEventElapsedTime(&msecTotal, start, stop);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }

    
		
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    else {
        // Compute and print the performance
        float msecPerVectorAdd = msecTotal / nIter;
        double opsPerVectorAdd = 1.0 * (double)size;
        double megaOps = (opsPerVectorAdd * 1.0e-6f) / (msecPerVectorAdd / 1000.0f);
        printf("Performance= %.4f Mops/s, Time= %.4f msec, Size= %.0f Ops\n",
            megaOps, msecPerVectorAdd, opsPerVectorAdd);
        printf("WorkgroupSize: Threads= %u, Blocks= %u, Total Threads= %u\n\n",
            threads, blocks, threads * blocks);
    }
    
    // With GPU kernel completed do garbage collection and free up allocated GPU memory.
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
